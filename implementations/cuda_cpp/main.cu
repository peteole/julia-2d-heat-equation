#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <memory>
#include <chrono>
// #include "discretize_pde.hpp"

#include "cuda_heat.cu"

std::unique_ptr<std::vector<std::vector<float>>> device_to_vector_of_vectors(float *U_dev, int N)
{
    float *U_host = (float *)malloc(N * N * sizeof(float));
    if (!U_host)
    {
        std::cout << "failed to allocate host memory\n";
    }
    hipError_t err = hipMemcpy(U_host, U_dev, N * N * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cout << "Failed to copy memory to host: " << hipGetErrorString(err);
    }

    std::unique_ptr<std::vector<std::vector<float>>> out = std::make_unique<std::vector<std::vector<float>>>(N, std::vector<float>(N, 0));

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            (*out)[i][j] = U_host[i * N + j];
        }
    }
    free(U_host);
    return out;
}

void discretize_heat_equation_cuda()
{
    auto N = 2000;
    float h = 1.0 / (N - 1);
    float dt = 0.000002;

    float *U;
    float *U_temp;
    size_t size = N * N * sizeof(float);
    hipError_t err1 = hipMalloc(&U, size);
    hipError_t err2 = hipMalloc(&U_temp, size);

    if (err1 != hipSuccess)
    {
        std::cout << "Failed to allocate memory on device: " << hipGetErrorString(err1);
    }
    if (err2 != hipSuccess)
    {
        std::cout << "Failed to allocate memory on device: " << hipGetErrorString(err2);
    }

    auto start = std::chrono::steady_clock::now();

    dim3 threadsPerBlock(8, 8);
    dim3 numBlocks = ((int)std::ceil((float)N / threadsPerBlock.x), (int)std::ceil((float)N / threadsPerBlock.y));

    domain_setup<<<numBlocks, threadsPerBlock>>>(U, N);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cout << "CUDA Error after domain_setup: " << hipGetErrorString(err) << std::endl;
    }

    int iteration = 1;
    for (double t = 0; t < 0.01; t += dt)
    {
        heat_step<<<numBlocks, threadsPerBlock>>>(U, U_temp, N, dt, h);
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            std::cout << "CUDA Error after heat_step: " << hipGetErrorString(err) << std::endl;
        }

        iteration++;

        float *temp = U_temp;
        U_temp = U;
        U = temp;
    }

    auto host_vector_of_vectors = device_to_vector_of_vectors(U_temp, N);

    hipFree(U);
    hipFree(U_temp);

    auto end = std::chrono::steady_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "U[10][10]" << (*host_vector_of_vectors)[10][10] << "\n";
    std::cout << "completed operation in " << duration << "ms";
    std::cout << "iterations: " << iteration << "\n";
}

int main(int argc, char **argv)
{
    discretize_heat_equation_cuda();
}