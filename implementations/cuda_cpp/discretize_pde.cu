#include <vector>
#include <iostream>
#include <memory>
#include <chrono>
#include <string>
#include <fstream>
// #include "discretize_pde.hpp"

#include "kernels.cu"

void write_solution(float *U_dev, int N, std::string path)
{
    float *U_host = (float *)malloc(N * N * sizeof(float));
    if (!U_host)
    {
        std::cout << "failed to allocate host memory\n";
    }
    hipError_t err = hipMemcpy(U_host, U_dev, N * N * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cout << "Failed to copy memory to host: " << hipGetErrorString(err);
    }
    // write to file
    std::cout << "writing to file: " << path << std::endl;
    std::ofstream file;
    file.open(path);
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            file << U_host[i * N + j] << " ";
        }
        file << "\n";
    }
    file.close();
    free(U_host);
}

void discretize_heat_equation_cuda(int N, float dt, float t_end, int write_every)
{
    float h = 1.0 / (N - 1);

    float *U;
    float *U_temp;
    size_t size = N * N * sizeof(float);
    hipError_t err1 = hipMalloc(&U, size);
    hipError_t err2 = hipMalloc(&U_temp, size);

    if (err1 != hipSuccess)
    {
        std::cout << "Failed to allocate memory on device: " << hipGetErrorString(err1);
    }
    if (err2 != hipSuccess)
    {
        std::cout << "Failed to allocate memory on device: " << hipGetErrorString(err2);
    }


    dim3 threadsPerBlock(8, 8);
    dim3 numBlocks(N / threadsPerBlock.x + 1, (N / threadsPerBlock.y + 1));

    domain_setup<<<numBlocks, threadsPerBlock>>>(U, N);
    domain_setup<<<numBlocks, threadsPerBlock>>>(U_temp, N);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cout << "CUDA Error after domain_setup: " << hipGetErrorString(err) << std::endl;
    }

    int iteration = 1;
    for (double t = 0; t < t_end; t += dt)
    {
        heat_step<<<numBlocks, threadsPerBlock>>>(U, U_temp, N, dt, h);
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            std::cout << "CUDA Error after heat_step: " << hipGetErrorString(err) << std::endl;
        }
        if (iteration % write_every == 0 && write_every > 0)
        {
            write_solution(U, N, "output_raw/" + std::to_string(iteration)+","+std::to_string(t) + ".txt");
        }

        iteration++;

        float *temp = U_temp;
        U_temp = U;
        U = temp;
    }


    hipFree(U);
    hipFree(U_temp);
}