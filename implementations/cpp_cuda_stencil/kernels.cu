
#include <hip/hip_runtime.h>
__global__ void domain_setup(float* U, int N){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(i > 0 && i < N - 1 && j > 0 && j < N - 1){
        U[i * N + j] = 1;
    }else{
        if(i < N && j < N){
            U[i * N + j] = 0;
        }
    }
}

__global__ void heat_step(float* U, float* U_temp, int N, float dt, float h){
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i > 0 && i < N - 1 && j > 0 && j < N - 1){
        U_temp[i * N + j] = U[i * N + j] + dt / (4 * h * h) * (U[(i - 1) * N + j] + U[(i + 1) * N + j] + U[i * N + j + 1] + U[i * N + j - 1] - 4 * U[i * N + j]);
    }
}



